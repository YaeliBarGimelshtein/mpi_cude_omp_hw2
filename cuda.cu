#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "cudaHeader.h"


__global__ void calculateHistByDevice(int* input, int* histogram)
{
    //CREATE HIST FOR EACH BLOCK
    __shared__ int private_hist[SIZE];

    //CREATE INDEX FOR EACH THREAD IN EACH BLOCK
    int index = threadIdx.x + blockIdx.x * blockDim.x ; 

    //INITIATE THE RESULTS
    private_hist[index] = 0;

    printf("check\n");
    
    //COMPUTE HIST FOR EACH BLOCK
    atomicAdd(&private_hist[input[index]], 1);
    
    //MERGE ALL PRIVATE HISTS INTO OUTPUT
    atomicAdd(&histogram[input[index]], private_hist[index]);
    //atomicAdd(&histogram[input[index]], 1);
    //output[index] += private_hist[index];
}




void calculateHistByCuda(int* input, int size_of_input, int* result)
{
    int num_blocks = size_of_input / NUM_THREADS_PER_BLOCK;
    if(size_of_input % NUM_THREADS_PER_BLOCK != 0)
        num_blocks++;

    /*
    //ALLOCATE DATA TO CUDA MEMORY
    int* cuda_input, *cuda_hist;
    hipMalloc((void**)&cuda_input, size_of_input);
    hipMalloc((void**)&cuda_hist, SIZE);
    
    //COPY INPUT INTO DEVICE
    hipMemcpy(cuda_input, input, size_of_input, hipMemcpyHostToDevice);
    hipMemcpy(cuda_hist, result, SIZE, hipMemcpyHostToDevice);
    */
    //LUNCH KERNEL
    //calculateHistByCuda<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(input, result);

    calculateHistByDevice<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(input, result);
    

    //COPY RESULT BACK TO HOST
    //hipMemcpy(result, cuda_hist, SIZE, hipMemcpyDeviceToHost);
    
    //FREE
    //hipFree(cuda_input);
    //hipFree(cuda_hist);
    
    printf("The histogram from cuda:\n");
    for (int i = 0; i < SIZE; i++)
    {
        if(result[i] != 0)
            printf("%d : %d\n",i, result[i]); 
    }
}